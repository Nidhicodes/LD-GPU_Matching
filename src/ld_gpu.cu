#include "hip/hip_runtime.h"
#include "../include/ld_gpu.cuh"
#include <hip/hip_runtime.h>
#include <nccl.h>
#include <iostream>
#include <cassert>

// CUDA error checking macro
#define CUDA_CHECK(call)                                                          \
    do                                                                            \
    {                                                                             \
        hipError_t error = call;                                                 \
        if (error != hipSuccess)                                                 \
        {                                                                         \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << " - " \
                      << hipGetErrorString(error) << std::endl;                  \
            exit(1);                                                              \
        }                                                                         \
    } while (0)

// NCCL error checking macro
#define NCCL_CHECK(call)                                                          \
    do                                                                            \
    {                                                                             \
        ncclResult_t error = call;                                                \
        if (error != ncclSuccess)                                                 \
        {                                                                         \
            std::cerr << "NCCL error at " << __FILE__ << ":" << __LINE__ << " - " \
                      << ncclGetErrorString(error) << std::endl;                  \
            exit(1);                                                              \
        }                                                                         \
    } while (0)

    __device__ size_t warpReduceMax(size_t val, float weight, float *max_weight)
    {
        for (int offset = 16; offset > 0; offset /= 2)
        {
            float other_weight = __shfl_down_sync(0xffffffff, weight, offset);
            size_t other_val = __shfl_down_sync(0xffffffff, val, offset);
    
            if (other_weight > weight)
            {
                weight = other_weight;
                val = other_val;
            }
        }
        *max_weight = weight;
        return val;
    }

// Kernel for the pointing phase
__global__ void setPointersKernel(size_t *vertex_batch, size_t num_vertices_batch,
                               size_t *offsets, size_t *edges, float *weights,
                               size_t *pointers, size_t *mate,
                               size_t gpu_vertex_offset)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_vertices_batch)
        return;

    size_t local_u = vertex_batch[idx];  // Local vertex ID within partition
    size_t global_u = local_u + gpu_vertex_offset; // Global vertex ID

    // Skip if already matched
    if (mate[global_u] != SIZE_MAX)
        return;

    size_t best_v = SIZE_MAX;
    float best_weight = -1.0f;

    // Process neighbors
    size_t start = offsets[u];
    size_t end = offsets[u + 1];

    for (size_t e = start; e < end; ++e)
    {
        size_t v = edges[e];

        // Skip if already matched
        if (mate[v] != SIZE_MAX)
            continue;

        float w = weights[e];
        if (w > best_weight)
        {
            best_weight = w;
            best_v = v;
        }
    }

    float thread_best_weight = best_weight;
    size_t thread_best_v = best_v;
    // Use warp reduction
    best_v = warpReduceMax(thread_best_v, thread_best_weight, &best_weight);

    // Set pointer to heaviest available neighbor
    pointers[global_u] = best_v;
}

// Kernel for the matching phase
__global__ void setMatesKernel(size_t *vertex_batch, size_t num_vertices_batch,
                               size_t *pointers, size_t *mate,
                               size_t gpu_vertex_offset, int *has_new_matches)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_vertices_batch)
        return;

    size_t local_u = vertex_batch[idx];  // Local vertex ID within partition
    size_t global_u = local_u + gpu_vertex_offset; // Global vertex ID

    // Skip if already matched
    if (mate[global_u] != SIZE_MAX)
        return;

    size_t v = pointers[global_u];

    // Check for mutual pointing
    if (v != SIZE_MAX && pointers[v] == global_u)
    {
        // Atomic operation to ensure only one thread sets the match
        if (atomicCAS(&mate[global_u], SIZE_MAX, v) == SIZE_MAX)
        {
            atomicCAS(&mate[v], SIZE_MAX, global_u);
            *has_new_matches = 1; // Mark that new matches were found
        }
    }
}

// Kernel to check for new matches
__global__ void checkNewMatchesKernel(size_t num_vertices, size_t* current_mate, size_t* old_mate, int* has_new_matches) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_vertices) return;
    
    if (current_mate[idx] != old_mate[idx]) {
        *has_new_matches = 1;
    }
}

// LD_GPU_Matcher implementation
LD_GPU_Matcher::LD_GPU_Matcher(Graph &graph, int num_gpus, int max_batches_per_device)
    : num_gpus(num_gpus), threads_per_block(256)
{
    // Initialize host arrays
    h_pointers.resize(graph.num_vertices, SIZE_MAX);
    h_mate.resize(graph.num_vertices, SIZE_MAX);

    // Set up devices
    std::cerr << "*** Requested " << num_gpus << " GPU(s), and ";
    setupDevices(num_gpus);

    std::cerr << num_gpus << " GPU(s) are available. ***" << std::endl;
    std::cout << "Using " << num_gpus << " GPU(s) for matching..." << std::endl
              << std::endl;

    // Partition graph
    graph.partitionGraph(num_gpus, graph_partitions);

    // Create batches
    createBatches(max_batches_per_device);

    // Set up NCCL for multi-GPU communication
    if (num_gpus > 1)
    {
        setupNCCL();
    }

    // Allocate memory on each device
    d_pointers.resize(num_gpus, nullptr);
    d_mate.resize(num_gpus, nullptr);

    for (int gpu = 0; gpu < num_gpus; ++gpu)
    {
        CUDA_CHECK(hipSetDevice(gpu));

        // Copy graph partition to device
        graph_partitions[gpu].copyToDevice();

        // Allocate pointers and mate arrays on device
        CUDA_CHECK(hipMalloc(&d_pointers[gpu], sizeof(size_t) * graph.num_vertices));
        CUDA_CHECK(hipMalloc(&d_mate[gpu], sizeof(size_t) * graph.num_vertices));

        // Initialize arrays
        CUDA_CHECK(hipMemset(d_pointers[gpu], 0xFF, sizeof(size_t) * graph.num_vertices)); // Set to SIZE_MAX
        CUDA_CHECK(hipMemset(d_mate[gpu], 0xFF, sizeof(size_t) * graph.num_vertices));     // Set to SIZE_MAX

        // Create streams
        streams[0].resize(num_gpus);
        streams[1].resize(num_gpus);
        for (int s = 0; s < 2; ++s)
        {
            CUDA_CHECK(hipStreamCreate(&streams[s][gpu]));
        }
    }
}

LD_GPU_Matcher::~LD_GPU_Matcher()
{
    // Clean up device memory
    for (int gpu = 0; gpu < num_gpus; ++gpu)
    {
        CUDA_CHECK(hipSetDevice(gpu));

        if (d_pointers[gpu])
        {
            CUDA_CHECK(hipFree(d_pointers[gpu]));
        }

        if (d_mate[gpu])
        {
            CUDA_CHECK(hipFree(d_mate[gpu]));
        }

        // Free graph partition memory
        graph_partitions[gpu].freeDeviceMemory();

        // Destroy streams
        for (int s = 0; s < 2; ++s)
        {
            CUDA_CHECK(hipStreamDestroy(streams[s][gpu]));
        }
    }

    // Clean up NCCL
    if (num_gpus > 1)
    {
        cleanupNCCL();
    }
}

void LD_GPU_Matcher::setupDevices(int &num_gpus)
{
    int device_count;
    CUDA_CHECK(hipGetDeviceCount(&device_count));

    if (num_gpus > device_count)
    {
        num_gpus = device_count;
    }
}

void LD_GPU_Matcher::createBatches(int max_batches_per_device) {
    batch_offsets.resize(num_gpus);
    
    // Calculate vertex offsets for each GPU
    gpu_vertex_offsets.resize(num_gpus, 0);
    for (int gpu = 1; gpu < num_gpus; ++gpu) {
        gpu_vertex_offsets[gpu] = gpu_vertex_offsets[gpu-1] + graph_partitions[gpu-1].num_vertices;
    }
    
    for (int gpu = 0; gpu < num_gpus; ++gpu) {
        const Graph& partition = graph_partitions[gpu];
        size_t vertices_per_batch = (partition.num_vertices + max_batches_per_device - 1) / max_batches_per_device;
        
        std::vector<size_t>& gpu_batches = batch_offsets[gpu];
        gpu_batches.clear();
        gpu_batches.push_back(0);
        
        for (size_t v = vertices_per_batch; v < partition.num_vertices; v += vertices_per_batch) {
            gpu_batches.push_back(v);
        }
        
        gpu_batches.push_back(partition.num_vertices);
        
        std::cout << "// GPU " << gpu << " has " << (gpu_batches.size() - 1) << " batches. //" << std::endl;
    }
    std::cout << std::endl;
}

void LD_GPU_Matcher::setupNCCL()
{
    comms.resize(num_gpus);
    int devices[num_gpus];
    for (int i = 0; i < num_gpus; ++i)
    {
        devices[i] = i;
    }

    // Initialize NCCL communicators
    NCCL_CHECK(ncclCommInitAll(comms.data(), num_gpus, devices));
}

void LD_GPU_Matcher::cleanupNCCL()
{
    for (int i = 0; i < num_gpus; ++i)
    {
        ncclCommDestroy(comms[i]);
    }
}

bool LD_GPU_Matcher::executeIterationBatched() {
    int has_new_matches = 0;
    
    // Allocate device memory for the has_new_matches flag on GPU 0
    int* d_has_new_matches;
    CUDA_CHECK(hipSetDevice(0));
    CUDA_CHECK(hipMalloc(&d_has_new_matches, sizeof(int)));
    CUDA_CHECK(hipMemset(d_has_new_matches, 0, sizeof(int)));
    
    // Save old matching for comparison
    std::vector<size_t> old_mate = h_mate;
    
    // Sync host-to-device for all GPUs at the start
    for (int gpu = 0; gpu < num_gpus; ++gpu) {
        CUDA_CHECK(hipSetDevice(gpu));
        CUDA_CHECK(hipMemcpy(d_mate[gpu], h_mate.data(), sizeof(size_t) * h_mate.size(), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_pointers[gpu], h_pointers.data(), sizeof(size_t) * h_pointers.size(), hipMemcpyHostToDevice));
    }
    
    // Pointing phase
    for (int gpu = 0; gpu < num_gpus; ++gpu) {
        CUDA_CHECK(hipSetDevice(gpu));
        
        const Graph& partition = graph_partitions[gpu];
        const std::vector<size_t>& gpu_batches = batch_offsets[gpu];
        size_t gpu_offset = gpu_vertex_offsets[gpu];
        
        // Process each batch
        for (size_t b = 0; b < gpu_batches.size() - 1; ++b) {
            int stream_idx = b % 2;
            hipStream_t& stream = streams[stream_idx][gpu];
            
            size_t batch_start = gpu_batches[b];
            size_t batch_end = gpu_batches[b + 1];
            size_t batch_size = batch_end - batch_start;
            
            // Create batch of local vertex IDs within the partition
            std::vector<size_t> vertex_batch(batch_size);
            for (size_t i = 0; i < batch_size; ++i) {
                size_t global_vertex_id = batch_start + i;
                if (gpu > 0) {
                    size_t offset = 0;
                    for (int prev_gpu = 0; prev_gpu < gpu; ++prev_gpu) {
                        offset += graph_partitions[prev_gpu].num_vertices;
                    }
                    global_vertex_id += offset;
                }
                vertex_batch[i] = global_vertex_id;
            }
            
            // Allocate memory for batch on device
            size_t* d_vertex_batch;
            CUDA_CHECK(hipMalloc(&d_vertex_batch, sizeof(size_t) * batch_size));
            CUDA_CHECK(hipMemcpyAsync(d_vertex_batch, vertex_batch.data(), sizeof(size_t) * batch_size, 
                                      hipMemcpyHostToDevice, stream));
            
            // Launch kernel for pointing phase with GPU offset
            int blocks = (batch_size + threads_per_block - 1) / threads_per_block;
            setPointersKernel<<<blocks, threads_per_block, 0, stream>>>(
                d_vertex_batch, batch_size,
                partition.d_offsets, partition.d_edges, partition.d_weights,
                d_pointers[gpu], d_mate[gpu], gpu_offset
            );
            
            // Free batch memory
            CUDA_CHECK(hipFree(d_vertex_batch));
            
            // Synchronize stream
            CUDA_CHECK(hipStreamSynchronize(stream));
        }
    }
    
    // All devices synchronize pointers via NCCL if multi-GPU
    if (num_gpus > 1) {
        for (int gpu = 0; gpu < num_gpus; ++gpu) {
            CUDA_CHECK(hipSetDevice(gpu));
            NCCL_CHECK(ncclAllReduce(
                (const void*)d_pointers[gpu], (void*)d_pointers[gpu], 
                h_pointers.size(), ncclUint64, ncclMax, 
                comms[gpu], streams[0][gpu]));
        }
        
        // Synchronize all streams
        for (int gpu = 0; gpu < num_gpus; ++gpu) {
            CUDA_CHECK(hipSetDevice(gpu));
            CUDA_CHECK(hipStreamSynchronize(streams[0][gpu]));
        }
    }
    
    // Matching phase - each GPU works on its assigned partition
    for (int gpu = 0; gpu < num_gpus; ++gpu) {
        CUDA_CHECK(hipSetDevice(gpu));
        
        const Graph& partition = graph_partitions[gpu];
        const std::vector<size_t>& gpu_batches = batch_offsets[gpu];
        size_t gpu_offset = gpu_vertex_offsets[gpu];
        
        // Process each batch
        for (size_t b = 0; b < gpu_batches.size() - 1; ++b) {
            int stream_idx = b % 2;
            hipStream_t& stream = streams[stream_idx][gpu];
            
            size_t batch_start = gpu_batches[b];
            size_t batch_end = gpu_batches[b + 1];
            size_t batch_size = batch_end - batch_start;
            
            // Create batch of local vertex IDs
            std::vector<size_t> vertex_batch(batch_size);
            for (size_t i = 0; i < batch_size; ++i) {
                vertex_batch[i] = batch_start + i;
            }
            
            // Allocate memory for batch on device
            size_t* d_vertex_batch;
            CUDA_CHECK(hipMalloc(&d_vertex_batch, sizeof(size_t) * batch_size));
            CUDA_CHECK(hipMemcpyAsync(d_vertex_batch, vertex_batch.data(), sizeof(size_t) * batch_size, 
                                      hipMemcpyHostToDevice, stream));
            
            // Launch kernel for matching phase
            int blocks = (batch_size + threads_per_block - 1) / threads_per_block;
            setMatesKernel<<<blocks, threads_per_block, 0, stream>>>(
                d_vertex_batch, batch_size,
                d_pointers[gpu], d_mate[gpu], gpu_offset, d_has_new_matches
            );
            
            // Free batch memory
            CUDA_CHECK(hipFree(d_vertex_batch));
            
            // Synchronize stream
            CUDA_CHECK(hipStreamSynchronize(stream));
        }
    }
    
    // Synchronize mates across GPUs if multi-GPU
    if (num_gpus > 1) {
        for (int gpu = 0; gpu < num_gpus; ++gpu) {
            CUDA_CHECK(hipSetDevice(gpu));
            NCCL_CHECK(ncclAllReduce(
                (const void*)d_mate[gpu], (void*)d_mate[gpu], 
                h_mate.size(), ncclUint64, ncclMin, 
                comms[gpu], streams[0][gpu]));
            
            // Also sync the has_new_matches flag
            if (gpu > 0) {
                int* gpu_has_new_matches;
                CUDA_CHECK(hipMalloc(&gpu_has_new_matches, sizeof(int)));
                NCCL_CHECK(ncclBroadcast(
                    (const void*)d_has_new_matches, (void*)gpu_has_new_matches, 
                    sizeof(int), ncclInt, 0, comms[gpu], streams[0][gpu]));
                CUDA_CHECK(hipFree(gpu_has_new_matches));
            }
        }
        
        // Synchronize all streams
        for (int gpu = 0; gpu < num_gpus; ++gpu) {
            CUDA_CHECK(hipSetDevice(gpu));
            CUDA_CHECK(hipStreamSynchronize(streams[0][gpu]));
        }
    }
    
    // Copy updated matching back to host from GPU 0
    CUDA_CHECK(hipSetDevice(0));
    CUDA_CHECK(hipMemcpy(h_mate.data(), d_mate[0], sizeof(size_t) * h_mate.size(), hipMemcpyDeviceToHost));
    
    // Copy updated pointers back to host
    CUDA_CHECK(hipMemcpy(h_pointers.data(), d_pointers[0], sizeof(size_t) * h_pointers.size(), hipMemcpyDeviceToHost));
    
    // Copy has_new_matches flag
    CUDA_CHECK(hipMemcpy(&has_new_matches, d_has_new_matches, sizeof(int), hipMemcpyDeviceToHost));
    
    // If no matches found by direct flag, check by comparing old and new matches
    if (has_new_matches == 0) {
        for (size_t i = 0; i < h_mate.size(); ++i) {
            if (h_mate[i] != old_mate[i]) {
                has_new_matches = 1;
                break;
            }
        }
    }
    
    // Count current matched pairs
    size_t matched_count = 0;
    for (size_t i = 0; i < h_mate.size(); ++i) {
        if (h_mate[i] != SIZE_MAX && i < h_mate[i]) {
            matched_count++;
        }
    }
    std::cout << "     Total matched pairs: " << matched_count << std::endl;
    
    // Free device memory
    CUDA_CHECK(hipFree(d_has_new_matches));
    
    std::cout << "==> New matches found: " << (has_new_matches == 1 ? "yes" : "no") << std::endl;
    
    return has_new_matches == 1;
}

void LD_GPU_Matcher::computeMatching() {
    int iteration = 0;
    bool continue_matching = true;
    
    // Initialize pointers to SIZE_MAX
    std::fill(h_pointers.begin(), h_pointers.end(), SIZE_MAX);
    
    while (continue_matching) {
        std::cout << "  [Starting iteration " << iteration << "]" << std::endl;
        
        continue_matching = executeIterationBatched();
        
        if (!continue_matching) {
            std::cout << "==> No new matches found in iteration " << iteration << std::endl << std::endl;
            break;
        }
        
        std::cout << std::endl;
        iteration++;
    }
    
    // Count matches
    size_t num_matches = 0;
    for (size_t i = 0; i < h_mate.size(); ++i) {
        if (h_mate[i] != SIZE_MAX && i < h_mate[i]) {
            num_matches++;
        }
    }
    
    std::cout << ">>>> Matching completed. <<<<" << std::endl;
    std::cout << "\n\n# Final Results:" << std::endl;
    std::cout << "- Matching completed in " << iteration << " iterations." << std::endl;
    std::cout << "- Found " << num_matches << " matched pairs." << std::endl;
}

const std::vector<size_t> &LD_GPU_Matcher::getMatching() const
{
    return h_mate;
}